#include "hip/hip_runtime.h"
#ifndef NEMO_CUDA_PLUGINS_IZHIKEVICH_CU
#define NEMO_CUDA_PLUGINS_IZHIKEVICH_CU

/* Copyright 2010 Imperial College London
 *
 * This file is part of NeMo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

/*! \file Izhikevich.cu Izhikevich neuron update kernel */

#include <nemo/config.h>
#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
#	include <log.cu_h>
#endif
#include <bitvector.cu>
#include <current.cu>
#include <firing.cu>
#include <neurons.cu>
#include <parameters.cu>
#include <rng.cu>

#include <nemo/plugins/Izhikevich.h>
#include "neuron_model.h"



/*! Update state of all neurons
 *
 * Update the state of all neurons in partition according to the equations in
 * Izhikevich's 2003 paper based on
 *
 * - the neuron parameters (a-d)
 * - the neuron state (u, v)
 * - input current (from other neurons, random input current, or externally provided)
 * - per-neuron specific firing stimulus
 *
 * The neuron state is updated using the Euler method.
 *
 * \param[in] s_partitionSize
 *		number of neurons in current partition
 * \param[in] g_neuronParameters
 *		global memory containing neuron parameters (see \ref nemo::cuda::Neurons)
 * \param[in] g_neuronState
 *		global memory containing neuron state (see \ref nemo::cuda::Neurons)
 * \param[in] s_current
 *		shared memory vector containing input current for all neurons in
 *		partition
 * \param[in] s_fstim
 *		shared memory bit vector where set bits indicate neurons which should
 *		be forced to fire
 * \param[out] s_nFired
 *		output variable which will be set to the number of	neurons which fired
 *		this cycle
 * \param[out] s_fired
 *		shared memory vector containing local indices of neurons which fired.
 *		s_fired[0:s_nFired-1] will contain valid data, whereas remaining
 *		entries may contain garbage.
 */
__device__
void
updateNeurons(
	uint32_t cycle,
	const param_t& s_params,
	unsigned globalPartitionCount,
	unsigned s_globalPartition,
	unsigned s_partitionSize,
	float* g_neuronParameters,
	float* g_neuronState,
	uint32_t* s_valid,   // bitvector for valid neurons
	// input
	nrng_t g_nrng,
	float* g_currentE,
	float* g_currentI,
	float* s_currentExt,    // external input current
	// buffers
	uint32_t* s_fstim,
	// output
	unsigned* s_nFired,
	nidx_dt* s_fired)    // s_NIdx, so can handle /all/ neurons firing
{
	//! \todo could set these in shared memory
	size_t neuronParametersSize = PARTITION_COUNT * s_params.pitch32;
	const float* g_a = g_neuronParameters + PARAM_A * neuronParametersSize;
	const float* g_b = g_neuronParameters + PARAM_B * neuronParametersSize;
	const float* g_c = g_neuronParameters + PARAM_C * neuronParametersSize;
	const float* g_d = g_neuronParameters + PARAM_D * neuronParametersSize;
	const float* g_sigma = g_neuronParameters + PARAM_SIGMA * neuronParametersSize;

	//! \todo avoid repeated computation of the same data here
	const float* g_u0 = state<1, 2, STATE_U>(cycle, s_params.pitch32, g_neuronState);
	const float* g_v0 = state<1, 2, STATE_V>(cycle, s_params.pitch32, g_neuronState);
	float* g_u1 = state<1, 2, STATE_U>(cycle+1, s_params.pitch32, g_neuronState);
	float* g_v1 = state<1, 2, STATE_V>(cycle+1, s_params.pitch32, g_neuronState);

	for(unsigned nbase=0; nbase < s_partitionSize; nbase += THREADS_PER_BLOCK) {

		unsigned neuron = nbase + threadIdx.x;

		/* if index space is contigous, no warp divergence here */
		if(bv_isSet(neuron, s_valid)) {

			float v = g_v0[neuron];
			float u = g_u0[neuron];
			float a = g_a[neuron];
			float b = g_b[neuron];

			float I = g_currentE[neuron] + g_currentI[neuron] + s_currentExt[neuron];

			float sigma = g_sigma[neuron];
			if(sigma != 0.0f) {
				I += nrand(globalPartitionCount, s_globalPartition, neuron, g_nrng) * sigma;
			}

			/* n sub-steps for numerical stability, with u held */
			bool fired = false;
			for(int j=0; j < 4; ++j) {
				if(!fired) { 
					v += 0.25f * ((0.04f*v + 5.0f) * v + 140.0f - u + I);
					/*! \todo: could pre-multiply this with a, when initialising memory */
					u += 0.25f * (a * ( b*v - u ));
					fired = v >= 30.0f;
				} 
			}

			bool forceFiring = bv_isSet(neuron, s_fstim); // (smem broadcast)

			if(fired || forceFiring) {

				/* Only a subset of the neurons fire and thus require c/d
				 * fetched from global memory. One could therefore deal with
				 * all the fired neurons separately. This was found, however,
				 * to slow down the fire step by 50%, due to extra required
				 * synchronisation.  */
				v = g_c[neuron];
				u += g_d[neuron];

#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
				DEBUG_MSG("c%u ?+%u-%u fired (forced: %u)\n",
						s_cycle, CURRENT_PARTITION, neuron, forceFiring);
#endif
				unsigned i = atomicAdd(s_nFired, 1);
				s_fired[i] = neuron;
			}

			g_v1[neuron] = v;
			g_u1[neuron] = u;
		}

		__syncthreads();
	}
}



/*! Update the state of all Izhikevich neurons in the network */
__global__
void
updateNeurons(
		uint32_t cycle,
		unsigned globalPartitionCount,
		unsigned basePartition,
		unsigned* g_partitionSize,
		param_t* g_params,
		// neuron state
		float* gf_neuronParameters,
		float* gf_neuronState,
		nrng_t g_nrng,
		uint32_t* g_valid,
		// firing stimulus
		uint32_t* g_fstim,
		float* g_istim,
		float* g_current,
		uint32_t* g_firingOutput, // dense output, already offset to current cycle
		unsigned* g_nFired,       // device-only buffer
		nidx_dt* g_fired)         // device-only buffer, sparse output
{
	__shared__ nidx_dt s_fired[MAX_PARTITION_SIZE];

	__shared__ unsigned s_nFired;
	__shared__ unsigned s_partitionSize;

	__shared__ unsigned s_globalPartition;

	if(threadIdx.x == 0) {
#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
		s_cycle = cycle;
#endif
		s_nFired = 0;
		s_globalPartition = basePartition + CURRENT_PARTITION;
		s_partitionSize = g_partitionSize[s_globalPartition];
    }
	__syncthreads();

	__shared__ param_t s_params;
	loadParameters(g_params, &s_params);

	float* g_currentE = incomingExcitatory(g_current, globalPartitionCount, s_globalPartition, s_params.pitch32);
	float* g_currentI = incomingInhibitory(g_current, globalPartitionCount, s_globalPartition, s_params.pitch32);

	__shared__ float s_current[MAX_PARTITION_SIZE];
	loadCurrentStimulus(s_globalPartition, s_partitionSize, s_params.pitch32, g_istim, s_current);

	__shared__ uint32_t s_fstim[S_BV_PITCH];
	loadFiringInput(s_globalPartition, s_params.pitch1, g_fstim, s_fstim);

	__shared__ uint32_t s_valid[S_BV_PITCH];
	bv_copy(g_valid + CURRENT_PARTITION * s_params.pitch1, s_valid);
	__syncthreads();

	updateNeurons(
			cycle,
			s_params,
			globalPartitionCount,
			s_globalPartition,
			s_partitionSize,
			//! \todo use consistent parameter passing scheme here
			gf_neuronParameters + CURRENT_PARTITION * s_params.pitch32,
			gf_neuronState,
			s_valid,
			g_nrng,
			g_currentE, g_currentI,
			s_current, s_fstim,
			&s_nFired,
			s_fired);

	__syncthreads();

	storeDenseFiring(s_nFired, s_globalPartition, s_params.pitch1, s_fired, g_firingOutput);
	storeSparseFiring(s_nFired, s_globalPartition, s_params.pitch32, s_fired, g_nFired, g_fired);
}



/*! Wrapper for the __global__ call that performs a single simulation step */
extern "C"
NEMO_PLUGIN_DLL_PUBLIC
hipError_t
cuda_update_neurons(
		hipStream_t stream,
		unsigned cycle,
		unsigned globalPartitionCount,
		unsigned localPartitionCount,
		unsigned basePartition,
		unsigned* d_partitionSize,
		param_t* d_params,
		float* df_neuronParameters,
		float* df_neuronState,
		nrng_t d_nrng,
		uint32_t* d_valid,
		uint32_t* d_fstim,
		float* d_istim,
		float* d_current,
		uint32_t* d_fout,
		unsigned* d_nFired,
		nidx_dt* d_fired,
		struct rcm_dt* /* unused */)
{
	dim3 dimBlock(THREADS_PER_BLOCK);
	dim3 dimGrid(localPartitionCount);

	updateNeurons<<<dimGrid, dimBlock, 0, stream>>>(
			cycle, globalPartitionCount, basePartition,
			d_partitionSize, d_params,
			df_neuronParameters, df_neuronState, d_nrng, d_valid,
			d_fstim,   // firing stimulus
			d_istim,   // current stimulus
			d_current, // internal input current
			d_fout, d_nFired, d_fired);

	return hipGetLastError();
}

cuda_update_neurons_t* test_update = &cuda_update_neurons;

#include "default_init.c"

#endif
