#include "hip/hip_runtime.h"
#ifndef NEMO_CUDA_PLUGINS_QIF_CU
#define NEMO_CUDA_PLUGINS_QIF_CU

/* Copyright 2010 Imperial College London
 *
 * This file is part of NeMo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

/*! \file QIF.cu Quadratic intergrate and fire neuron update kernel */

#include <nemo/config.h>
#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
#	include <log.cu_h>
#endif
#include <bitvector.cu>
#include <current.cu>
#include <firing.cu>
#include <fixedpoint.cu>
#include <neurons.cu>
#include <parameters.cu>
#include <rng.cu>


#include "neuron_model.h"

#include <nemo/internal_types.h>
#include <math.h>





/*! Update state of all neurons
 *
 * Update the state of all neurons in partition according to Hodgkin Huxley equations 
 *
 * - the neuron parameters ()
 * - the neuron state (v, n, m, h, dir)
 * - input current (from other neurons, random input current, or externally provided)
 * - per-neuron specific firing stimulus
 *
 * The neuron state is updated using the Euler method.
 *
 * \param[in] s_partitionSize
 *		number of neurons in current partition
 * \param[in] g_neuronParameters
 *		global memory containing neuron parameters (see \ref nemo::cuda::Neurons)
 * \param[in] g_neuronState
 *		global memory containing neuron state (see \ref nemo::cuda::Neurons)
 * \param[in] s_current
 *		shared memory vector containing input current for all neurons in
 *		partition
 * \param[in] s_fstim
 *		shared memory bit vector where set bits indicate neurons which should
 *		be forced to fire
 * \param[out] s_nFired
 *		output variable which will be set to the number of	neurons which fired
 *		this cycle
 * \param[out] s_fired
 *		shared memory vector containing local indices of neurons which fired.
 *		s_fired[0:s_nFired-1] will contain valid data, whereas remaining
 *		entries may contain garbage.
 */
__device__
void
updateNeurons(
	uint32_t cycle,
	const param_t& s_params,
	unsigned globalPartitionCount,
	unsigned s_globalPartition,
	unsigned s_partitionSize,
	float* g_neuronParameters,
	float* g_neuronState,
	uint32_t* s_valid,   // bitvector for valid neurons
	// input
	nrng_t g_nrng,
	float* g_currentE,
	float* g_currentI,
	float* s_currentExt,    // external input current
	// buffers
	uint32_t* s_fstim,
	// output
	unsigned* s_nFired,
	nidx_dt* s_fired)    // s_NIdx, so can handle /all/ neurons firing
{
	#define PARAM_DUMMY 0
	#define STATE_V 0
	#define STATE_N 1
	#define STATE_M 2
	#define STATE_H 3
	#define STATE_DIR 4
	

	//! \todo could set these in shared memory
	//! \todo avoid repeated computation of the same data here
	const float* g_v0 = state<1, 6, STATE_V>(cycle, s_params.pitch32, g_neuronState);
	float* g_v1 = state<1, 6, STATE_V>(cycle+1, s_params.pitch32, g_neuronState);
	const float* g_n0 = state<1, 6, STATE_N>(cycle, s_params.pitch32, g_neuronState);
	float* g_n1 = state<1, 6, STATE_N>(cycle+1, s_params.pitch32, g_neuronState);
	const float* g_m0 = state<1, 6, STATE_M>(cycle, s_params.pitch32, g_neuronState);
	float* g_m1 = state<1, 6, STATE_M>(cycle+1, s_params.pitch32, g_neuronState);
	const float* g_h0 = state<1, 6, STATE_H>(cycle, s_params.pitch32, g_neuronState);
	float* g_h1 = state<1, 6, STATE_H>(cycle+1, s_params.pitch32, g_neuronState);
	const float* g_dir0 = state<1, 6, STATE_DIR>(cycle, s_params.pitch32, g_neuronState);
	float* g_dir1 = state<1, 6, STATE_DIR>(cycle+1, s_params.pitch32, g_neuronState);
	

	float dt = 0.001f; // Simulation time increment
	float gNa = 120.0f;
	float gK = 36.0f;
	float gL = 0.3f;
	float ENa = 115.0f-65.0f;
	float EK = -12.0f-65.0f;
	float EL = 10.6f-65.0f;
	float C = 1.0f;
	float RevE = 0.0f;
	float RevI = -70.0f;
	int inc_max= (int)(1/dt);

	for(unsigned nbase=0; nbase < s_partitionSize; nbase += THREADS_PER_BLOCK) {

		unsigned neuron = nbase + threadIdx.x;

		/* if index space is contigous, no warp divergence here */
		if(bv_isSet(neuron, s_valid)) {

			float v = g_v0[neuron];
			float n = g_n0[neuron];
			float m = g_m0[neuron];
			float h = g_h0[neuron];
			float dir = g_dir0[neuron];
			
			float Excit = g_currentE[neuron];
			float Inhib = g_currentI[neuron];
 			float Exter = s_currentExt[neuron];
			
			// Update v and u using QIF model in increments of tau
			bool fired = false;
			for(int k=1; k<=inc_max; ++k)
			{  

				float I = (Excit*(RevE-v)) + (Inhib*((RevI-v)/-1)) + Exter;
								
				float alphan = (0.1f-0.01f*(v+65.0f))/(exp(1.0f-0.1f*(v+65.0f))-1.0f);
				float alpham = (2.5f-0.1f*(v+65.0f))/(exp(2.5f-0.1f*(v+65.0f))-1.0f);
				float alphah = 0.07f*exp(-(v+65.0f)/20.0f);

				float betan = 0.125f*exp(-(v+65.0f)/80.0f);
				float betam = 4.0f*exp(-(v+65.0f)/18.0f);
				float betah = 1.0f/(exp(3.0f-0.1f*(v+65.0f))+1.0f);


				m = m + dt*(alpham*(1.0f-m)-betam*m);
				n = n + dt*(alphan*(1.0f-n)-betan*n);
				h = h + dt*(alphah*(1.0f-h)-betah*h);

				float Ik = gNa*(m*m*m)*h*(v-ENa) + gK*(n*n*n*n)*(v-EK) + gL*(v-EL);


				float newv = v + dt*(-Ik+I)/C;

				float new_dir = (newv-v);
				float change = dir<0 | newv<-45 ? 0 : new_dir;
				dir = new_dir;

				if(!fired && cycle >= 10)
					fired = change<0;
				
				v=newv;
			   	
			}      
			



			bool forceFiring = bv_isSet(neuron, s_fstim); // (smem broadcast)

			if(fired || forceFiring) {

				/* Only a subset of the neurons fire and thus require c/d
				 * fetched from global memory. One could therefore deal with
				 * all the fired neurons separately. This was found, however,
				 * to slow down the fire step by 50%, due to extra required
				 * synchronisation.  */
				//! \todo could probably hard-code c
				
#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
				DEBUG_MSG("c%u ?+%u-%u fired (forced: %u)\n",
						s_cycle, CURRENT_PARTITION, neuron, forceFiring);
#endif

				//! \todo consider *only* updating this here, and setting u and v separately
				unsigned i = atomicAdd(s_nFired, 1);

				/* can overwrite current as long as i < neuron. See notes below
				 * on synchronisation and declaration of s_current/s_fired. */
				s_fired[i] = neuron;
			}

		
			g_v1[neuron] = v;
			g_n1[neuron] = n;
			g_m1[neuron] = m;
			g_h1[neuron] = h;
			g_dir1[neuron] = dir;
		
		}

		/* synchronise to ensure accesses to s_fired and s_current (which use
		 * the same underlying buffer) do not overlap. Even in the worst case
		 * (all neurons firing) the write to s_fired will be at least one
		 * before the first unconsumed s_current entry. */
		__syncthreads();
	}
}



/*! Update the state of all neurons in the network */
__global__
void
updateNeurons(
		uint32_t cycle,
		unsigned globalPartitionCount,
		unsigned basePartition,
		unsigned* g_partitionSize,
		param_t* g_params,
		// neuron state
		float* gf_neuronParameters,
		float* gf_neuronState,
		nrng_t g_nrng,
		uint32_t* g_valid,
		// firing stimulus
		uint32_t* g_fstim,
		float* g_istim,
		float* g_current,
		uint32_t* g_firingOutput, // dense output, already offset to current cycle
		unsigned* g_nFired,       // device-only buffer
		nidx_dt* g_fired)         // device-only buffer, sparse output
{
	__shared__ nidx_dt s_fired[MAX_PARTITION_SIZE];

	__shared__ unsigned s_nFired;
	__shared__ unsigned s_partitionSize;

	__shared__ unsigned s_globalPartition;

	if(threadIdx.x == 0) {
#ifdef NEMO_CUDA_PLUGIN_DEBUG_TRACE
		s_cycle = cycle;
#endif
		s_nFired = 0;
		s_globalPartition = basePartition + CURRENT_PARTITION;
		s_partitionSize = g_partitionSize[s_globalPartition];
    }
	__syncthreads();

	__shared__ param_t s_params;
	loadParameters(g_params, &s_params);

	float* g_currentE = incomingExcitatory(g_current, globalPartitionCount, s_globalPartition, s_params.pitch32);
	float* g_currentI = incomingInhibitory(g_current, globalPartitionCount, s_globalPartition, s_params.pitch32);

	__shared__ float s_current[MAX_PARTITION_SIZE];
	loadCurrentStimulus(s_globalPartition, s_partitionSize, s_params.pitch32, g_istim, s_current);

	__shared__ uint32_t s_fstim[S_BV_PITCH];
	loadFiringInput(s_globalPartition, s_params.pitch1, g_fstim, s_fstim);

	__shared__ uint32_t s_valid[S_BV_PITCH];
	bv_copy(g_valid + CURRENT_PARTITION * s_params.pitch1, s_valid);
	__syncthreads();

	updateNeurons(
			cycle,
			s_params,
			globalPartitionCount,
			s_globalPartition,
			s_partitionSize,
			//! \todo use consistent parameter passing scheme here
			gf_neuronParameters + CURRENT_PARTITION * s_params.pitch32,
			gf_neuronState,
			s_valid,
			g_nrng,
			g_currentE, g_currentI,
			s_current, s_fstim,
			&s_nFired,
			s_fired);

	__syncthreads();

	storeDenseFiring(s_nFired, s_globalPartition, s_params.pitch1, s_fired, g_firingOutput);
	storeSparseFiring(s_nFired, s_globalPartition, s_params.pitch32, s_fired, g_nFired, g_fired);
}



/*! Wrapper for the __global__ call that performs a single simulation step */
extern "C"
NEMO_PLUGIN_DLL_PUBLIC
hipError_t
cuda_update_neurons(
		hipStream_t stream,
		unsigned cycle,
		unsigned globalPartitionCount,
		unsigned localPartitionCount,
		unsigned basePartition,
		unsigned* d_partitionSize,
		param_t* d_params,
		float* df_neuronParameters,
		float* df_neuronState,
		nrng_t d_nrng,
		uint32_t* d_valid,
		uint32_t* d_fstim,
		float* d_istim,
		float* d_current,
		uint32_t* d_fout,
		unsigned* d_nFired,
		nidx_dt* d_fired,
		struct rcm_dt* /* unused */)
{
	dim3 dimBlock(THREADS_PER_BLOCK);
	dim3 dimGrid(localPartitionCount);

	updateNeurons<<<dimGrid, dimBlock, 0, stream>>>(
			cycle, globalPartitionCount, basePartition,
			d_partitionSize, d_params,
			df_neuronParameters, df_neuronState, d_nrng, d_valid,
			d_fstim,   // firing stimulus
			d_istim,   // current stimulus
			d_current, // internal input current
			d_fout, d_nFired, d_fired);

	return hipGetLastError();
}


#include "default_init.c"

#endif
