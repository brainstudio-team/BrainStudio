#ifndef DEVICE_ASSERT_CU
#define DEVICE_ASSERT_CU

/* Copyright 2010 Imperial College London
 *
 * This file is part of nemo.
 *
 * This software is licenced for non-commercial academic use under the GNU
 * General Public Licence (GPL). You should have received a copy of this
 * licence along with nemo. If not, see <http://www.gnu.org/licenses/>.
 */

#include "kernel.cu_h"


#ifdef NEMO_CUDA_DEVICE_ASSERTIONS

#define DEVICE_ASSERTION_MEMSZ (MAX_PARTITION_COUNT * THREADS_PER_BLOCK)

__device__ uint32_t g_assertions[DEVICE_ASSERTION_MEMSZ];


__device__ __host__
size_t
assertion_offset(size_t partition, size_t thread)
{
    return partition * THREADS_PER_BLOCK + thread;
}



#ifdef __DEVICE_EMULATION__
#	define ASSERT(cond) assert(cond)
#else
#	define ASSERT(cond) \
        if(!(cond)) {\
			g_assertions[assertion_offset(CURRENT_PARTITION, threadIdx.x)] = __LINE__;\
        }
#endif
#else // NEMO_CUDA_DEVICE_ASSERTIONS
#   define ASSERT(cond)
#endif


__host__
hipError_t
getDeviceAssertions(unsigned partitions, uint32_t* h_assertions)
{
#ifdef NEMO_CUDA_DEVICE_ASSERTIONS
	size_t bytes = partitions * THREADS_PER_BLOCK * sizeof(uint32_t);
	return hipMemcpyFromSymbol(h_assertions, HIP_SYMBOL(g_assertions), bytes, hipMemcpyDeviceToHost);
#else
	return hipSuccess;
#endif
}


__host__
hipError_t
clearDeviceAssertions()
{
#ifdef NEMO_CUDA_DEVICE_ASSERTIONS
	void* addr;
	hipError_t err = hipGetSymbolAddress(&addr, HIP_SYMBOL(g_assertions));
	if(err != hipSuccess) {
		return err;
	}
	return hipMemset(addr, 0, DEVICE_ASSERTION_MEMSZ*sizeof(uint32_t));
#else
	return hipSuccess;
#endif
}

#endif
